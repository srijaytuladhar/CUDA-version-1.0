#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

/****************************************************************************

     Compilation Code:
     nvcc -o cuda_password_crack cuda_password_crack.cu
  
*****************************************************************************/


__device__ int is_a_match(char *attempt) {
  char plain_password1[] = "BV7842";
  char plain_password2[] = "ES2107";
  char plain_password3[] = "HR2332";
  char plain_password4[] = "RB9669";

  char *a = attempt;
  char *b = attempt;
  char *c = attempt;
  char *d = attempt;
  char *p1 = plain_password1;
  char *p2 = plain_password2;
  char *p3 = plain_password3;
  char *p4 = plain_password4;

  while(*a == *p1) {
   if(*a == '\0')
    {
    printf("%s\n",plain_password1);
      break;
    }

    a++;
    p1++;
  }
    
  while(*b == *p2) {
   if(*b == '\0')
    {
    printf("%s\n",plain_password2);
      break;
}

    b++;
    p2++;
  }

  while(*c == *p3) {
   if(*c == '\0')
    {
    printf("%s\n",plain_password3);
      break;
    }

    c++;
    p3++;
  }

  while(*d == *p4) {
   if(*d == '\0')
    {
    printf("%s",plain_password4);
      return 1;
    }

    d++;
    p4++;
  }
  return 0;

}
/****************************************************************************
  The kernel function assume that there will be only one thread and uses
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
char k1,k2,k3,k4;
 
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(k1='0'; k1<='9'; k1++){
      for(k2='0'; k2<='9'; k2++){
        for(k3='0'; k3<='9'; k3++){
          for(k4='0'; k4<='9'; k4++){
            password[2] = k1;
            password[3] = k2;
            password[4] = k3;
            password[5] = k4;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      }
   }
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);
  printf("\n===============================================================================\n");
  printf("!! MATCHED PASSWORD !! \n");
  printf("===============================================================================\n\n");
  kernel <<<26,26>>>();
  hipDeviceSynchronize();

  
  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("\n\n===============================================================================\n");
  printf("!! TIME TAKEN FOR EXECUTION !! \n");
  printf("===============================================================================\n\n");
  printf("Nanoseconds: %lld\n", time_elapsed); 
  printf("Seconds: %0.9lf\n", ((time_elapsed/1.0e9))); 
  printf("Minutes: %0.4lf\n", ((time_elapsed/1.0e9)/60));
  printf("Hours: %0.2lf\n\n", ((time_elapsed/1.0e9)/3600)); 
  

  return 0;
}



